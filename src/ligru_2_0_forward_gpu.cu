#include "hip/hip_runtime.h"
// Copyright 2022 Adel Moumen. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//    http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
// ==============================================================================

#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>
#include <stdexcept>

#include "blas.h"
#include "device_assert.h"

#include "inline_ops.h"
#include "layer_norm.h"
#include "ligru_2_0.h"

namespace {

template <typename T, haste::v0::ligru_2_0::ActivationFunction<T> activation, bool Training>
__global__ void PointwiseOperations(
    const int batch_dim,
    const int hidden_dim,
    const T *wx,
    const T *uh,
    const T *h,
    T *h_out,
    T *v
) {
  const int row = blockDim.x * blockIdx.x + threadIdx.x;
  const int col = blockDim.y * blockIdx.y + threadIdx.y;

  if (row >= hidden_dim || col >= batch_dim)
    return;

  const int weight_idx = col * (hidden_dim * 2) + row;
  const int output_idx = col * hidden_dim + row;

  const int a_idx = weight_idx + 0 * hidden_dim;
  const int z_idx = weight_idx + 1 * hidden_dim;

  const T z = sigmoid(wx[z_idx] + uh[z_idx]);
  const T a = wx[a_idx] + uh[z_idx];

  const T hcand = activation(a); 

  if (Training) {
    const int base_v_idx = col * (hidden_dim * 3) + row;
    v[base_v_idx + 1 * hidden_dim] = z;
    v[base_v_idx + 0 * hidden_dim] = a;
    v[base_v_idx + 2 * hidden_dim] = hcand;
  }

  T cur_h_value = z * h[output_idx] + (static_cast<T>(1.0) - z) * hcand;
  h_out[output_idx] = cur_h_value;
}
} // anonymous namespace

namespace haste {
namespace v0 {
namespace ligru_2_0 {

template <typename T> struct ForwardPass<T>::private_data {
  bool training;
  int batch_size;
  int input_size;
  int hidden_size;
  int activation;
  hipblasHandle_t blas_handle;
  hipStream_t stream[2];
  hipEvent_t event;
  hipStream_t sync_stream;
};

template <typename T>
ForwardPass<T>::ForwardPass(const bool training, const int batch_size,
                            const int input_size, const int hidden_size,
                            const hipblasHandle_t &blas_handle,
                            const int activation, const hipStream_t &stream)
    : data_(new private_data) {
  data_->activation = activation;
  data_->training = training;
  data_->batch_size = batch_size;
  data_->input_size = input_size;
  data_->hidden_size = hidden_size;
  data_->blas_handle = blas_handle;
  data_->sync_stream = stream;
  hipStreamCreate(&data_->stream[0]);
  hipStreamCreate(&data_->stream[1]);
  hipEventCreateWithFlags(&data_->event, hipEventDisableTiming);
}

template <typename T> ForwardPass<T>::~ForwardPass() {
  if (data_->sync_stream) {
    hipEventRecord(data_->event, data_->stream[1]);
    hipStreamWaitEvent(data_->sync_stream, data_->event, 0);
    hipEventRecord(data_->event, data_->stream[0]);
    hipStreamWaitEvent(data_->sync_stream, data_->event, 0);
  } else {
    hipStreamSynchronize(data_->stream[1]);
    hipStreamSynchronize(data_->stream[0]);
  }
  hipEventDestroy(data_->event);
  hipStreamDestroy(data_->stream[1]);
  hipStreamDestroy(data_->stream[0]);
  delete data_;
}

// returns a PointwiseOperations kernel given a runtime activation_method
template<typename T, bool training>
auto wrap_pointwise_activation(int activation_method)
{
    switch (activation_method)
    {
    case 0: return PointwiseOperations<T, relu, training>;
    case 1: return PointwiseOperations<T, leaky_relu, training>;
    case 2: return PointwiseOperations<T, sin, training>;
    case 3: return PointwiseOperations<T, tanh, training>;
    }

    throw std::invalid_argument("Invalid activation_method passed");
}

// returns a PointwiseOperations kernel given a runtime
// training+activation_method, by dispatching the proper value as a template
template<typename T>
auto wrap_pointwise(bool training, int activation_method)
{
    if (training)
    {
        return wrap_pointwise_activation<T, true>(activation_method);
    }
    else
    {
        return wrap_pointwise_activation<T, false>(activation_method);
    }
} 

template <typename T>
void ForwardPass<T>::IterateInternal(
    const T *u,
    const T *h,
    T *h_out,
    T *v,
    T *tmp_wx,
    T *tmp_uh,
    T *tmp_uh_norm,
    layer_norm::ForwardPass<T> &layer_norm1
) {
  const bool training = data_->training;
  const int batch_size = data_->batch_size;
  const int hidden_size = data_->hidden_size;
  const hipblasHandle_t blas_handle = data_->blas_handle;
  const hipStream_t stream1 = data_->stream[0];
  const hipEvent_t event = data_->event;

  hipblasSetStream(blas_handle, stream1);

  const static auto alpha = static_cast<T>(1.0f);
  const static auto beta = static_cast<T>(0.0f);
  // tmp_uh = 1 * u . h + 0 * tmp_uh aka tmp_uh = u.h
  // gemm is C <- alpha * A * B + beta * C
  blas<T>::gemm(
      blas_handle,
      HIPBLAS_OP_N, HIPBLAS_OP_N,                 // no transposition
      hidden_size * 2, batch_size, hidden_size, // shape
      &alpha,                                   // alpha == 1.0
      u, hidden_size * 2,                       // A
      h, hidden_size,                           // B
      &beta,                                    // beta == 0.0
      tmp_uh, hidden_size * 2                   // C, used only as a target
  );

  // NOTE: With FP16 inputs & outputs, LN will be performed in FP32 space internally
  layer_norm1.RunPartial(stream1, batch_size, tmp_uh, tmp_uh_norm);

  // Compute launch configuration for pointwise operations kernel.
  const dim3 blockDim(32, 16);
  const dim3 gridDim((hidden_size + blockDim.x - 1) / blockDim.x,
                     (batch_size + blockDim.y - 1) / blockDim.y);

  hipStreamWaitEvent(stream1, event, 0);

  const auto kernel = wrap_pointwise<T>(training, data_->activation);
  kernel<<<gridDim, blockDim, 0, stream1>>>(
    batch_size, hidden_size, tmp_wx, tmp_uh_norm, h, h_out, v
  );
}

template <typename T>
void ForwardPass<T>::Run(
    const int seq_length,
    T *wx, const T *u, T *h, T *v,
    layer_norm::ForwardPass<T> &layer_norm1,
    T *tmp_uh_norm,
    T *tmp_uh) {

  const blas<void>::set_pointer_mode scoped1(data_->blas_handle);

  const int batch_size = data_->batch_size;
  const int hidden_size = data_->hidden_size;
  const hipblasHandle_t blas_handle = data_->blas_handle;

  hipStream_t save_stream;
  hipblasGetStream(blas_handle, &save_stream);

  const int NH = batch_size * hidden_size;

  for (int i = 0; i < seq_length; ++i) {
    IterateInternal(u, h + i * NH, h + (i + 1) * NH, v + i * NH * 3,
                    wx + i * NH * 2, tmp_uh + i * NH * 2, tmp_uh_norm,
                    layer_norm1);
  }

  hipblasSetStream(blas_handle, save_stream);
}

template struct ForwardPass<half>;
template struct ForwardPass<float>;
template struct ForwardPass<double>;

} // namespace ligru_2_0
} // namespace v0
} // namespace haste